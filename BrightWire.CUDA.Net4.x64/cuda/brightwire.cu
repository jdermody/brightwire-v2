#include "hip/hip_runtime.h"
//Includes for IntelliSense 
#define _SIZE_T_DEFINED
#ifndef __HIPCC__
#define __HIPCC__
#endif
#ifndef __cplusplus
#define __cplusplus
#endif

#include <hip/hip_runtime.h>
#include <>
#include <>
#include "float.h"
#include <builtin_types.h>
#include <vector_functions.h>

#define BLOCKSIZE 16
#define BLOCKSIZE2 BLOCKSIZE*BLOCKSIZE

extern "C"
{
    __global__ void IsFinite(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = isfinite(a[index]) ? 0 : 1;
        }
	}

	__global__ void PointwiseMultiply(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] *= a[index];
        }
	}

	__global__ void PointwiseDivide(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = a[index] / b[index];
        }
	}

	__global__ void Sqrt(const float* __restrict a, float* __restrict b, int size, float valueAdjustment)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = sqrt(a[index] + valueAdjustment);
        }
	}

	__global__ void AddInPlace(float* __restrict a, const float* __restrict b, int size, float coefficient1, float coefficient2)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index] = (a[index] * coefficient1) + (b[index] * coefficient2);
        }
	}

	__global__ void SubtractInPlace(float* __restrict a, const float* __restrict b, int size, float coefficient1, float coefficient2)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index] = (a[index] * coefficient1) - (b[index] * coefficient2);
        }
	}

	__global__ void AddToEachRow(float* __restrict a, const float* __restrict b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                a[j * rows + i] += b[j];
            }
        }
	}

	__global__ void AddToEachColumn(float* __restrict a, const float* __restrict b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                a[j * rows + i] += b[i];
            }
        }
	}

	__global__ void TanH(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = tanh(a[index]);
        }
	}

	__global__ void TanHDerivative(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = 1.0f - pow(tanh(a[index]), 2);
        }
	}

	__global__ void Sigmoid(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[index] = 1.0f / (1.0f + exp(-1.0f * a[index]));
        }
	}

	__global__ void SigmoidDerivative(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float sigmoid = 1.0f / (1.0f + exp(-1.0f * a[index]));
			b[index] = sigmoid * (1.0f - sigmoid);
        }
	}

	__global__ void RELU(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0 : val;
        }
	}

	__global__ void RELUDerivative(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0 : 1;
        }
	}

	__global__ void LeakyRELU(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0.01f*val : val;
        }
	}

	__global__ void LeakyRELUDerivative(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = (val <= 0) ? 0.01f : 1;
        }
	}

	__global__ void Reverse(const float* __restrict a, float* __restrict b, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            b[size - index - 1] = a[index];
        }
	}

	__global__ void SumRows(const float* __restrict a, float* __restrict b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                atomicAdd(b + i, a[j * rows + i]);
            }
        }
	}

	__global__ void SumColumns(const float* __restrict a, float* __restrict b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                atomicAdd(b + j, a[j * rows + i]);
            }
        }
	}

	__global__ void MemClear(float* data, int count, int srcOffset, int srcIncrement)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            int arrayIndex = srcOffset + (index * srcIncrement);
			data[arrayIndex] = 0.0f;
        }
	}

	__global__ void FindMinAndMax(const float* __restrict data, int count, float* __restrict minBlock, float* __restrict maxBlock)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		block[tidX] = (index < count) ? data[index] : 0;
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float min = FLT_MAX, max = FLT_MIN;
			int maxIndex = BLOCKSIZE2;
			if (count - index < BLOCKSIZE2)
				maxIndex = count - index;
			for (int i = 0; i < maxIndex; i++) {
				float val = block[i];
				if (i == 0 || val > max)
					max = val;
				if (i == 0 || val < min)
					min = val;
			}
			minBlock[blockX] = min;
			maxBlock[blockX] = max;
		}
	}

	__global__ void FindSum(const float* __restrict data, int count, float* __restrict sum)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		if (index < count)
			block[tidX] = data[index];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			int maxIndex = BLOCKSIZE2;
			if (count - blockX * BLOCKSIZE2 < BLOCKSIZE2)
				maxIndex = count - blockX * BLOCKSIZE2;
			for (int i = 0; i < maxIndex; i++) {
				total += block[i];
			}
			sum[blockX] = total;
		}
	}

	__global__ void FindStdDev(const float* __restrict data, int count, float mean, float* __restrict stdDev)
	{
		int tidX = threadIdx.x;
		int blockX = blockIdx.x;
		int index = blockDim.x * blockX + tidX;

		// read block into shared memory
		__shared__ float block[BLOCKSIZE2];
		if (index < count)
			block[tidX] = data[index];
		__syncthreads();

		// aggregate per block
		if (tidX == 0) {
			float total = 0;
			int maxIndex = BLOCKSIZE2;
			if (count - blockX * BLOCKSIZE2 < BLOCKSIZE2)
				maxIndex = count - blockX * BLOCKSIZE2;
			for (int i = 0; i < maxIndex; i++) {
				total += pow(block[i] - mean, 2);
			}
			stdDev[blockX] = total;
		}
	}

	__global__ void Constrain(float* __restrict data, int count, float min, float max)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            float val = data[index];
			if (val < min)
				data[index] = min;
			if (val > max)
				data[index] = max;
        }
	}

	__global__ void Pow(const float* __restrict a, float* __restrict b, int count, float power)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            float val = a[index];
			b[index] = pow(val, power);
        }
	}

	__global__ void Diagonal(const float* __restrict a, float* __restrict b, int rows, int columns)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < rows && index < columns; index += blockDim.x * gridDim.x) {
            b[index] = a[index * rows + index];
        }
	}

	__global__ void L1Regularisation(float* __restrict a, int count, float coefficient)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            float val = a[index];
			a[index] = val - ((val > 0 ? 1 : val < 0 ? -1 : 0) * coefficient);
        }
	}

	__global__ void PointwiseDivideRows(float* __restrict a, const float* __restrict b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                int index = j * rows + i;
			    float val = a[index];
			    a[index] = val / b[i];
            }
        }
	}

	__global__ void PointwiseDivideColumns(float* __restrict a, const float* __restrict b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                int index = j * rows + i;
			    float val = a[index];
			    a[index] = val / b[j];
            }
        }
	}

	__global__ void SplitRows(const float* __restrict a, float* __restrict b, float* __restrict c, int rows, int columns, int position)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                int index = j * rows + i;
			    float val = a[index];
			    if(j >= position) {
				    int diff = j - position;
				    c[diff * rows + i] = val;
			    }else
				    b[index] = val;
            }
        }
	}

	__global__ void SplitColumns(const float* __restrict a, float* __restrict b, float* __restrict c, int rows, int columns, int position)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[j * rows + i];
			    if(i >= position) {
				    int diff = i - position;
				    c[j * (rows-position) + diff] = val;
			    }else
				    b[j * position + i] = val;
            }
        }
	}

	__global__ void ConcatColumns(const float* __restrict a, const float* __restrict b, float* __restrict c, int rows, int columns, int topRowCount, int bottomRowCount)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val;
			    if(i >= topRowCount)
				    val = b[j * bottomRowCount + i - topRowCount];
			    else
				    val = a[j * topRowCount + i];
			    c[j * rows + i] = val;
            }
        }
	}

	__global__ void ConcatRows(const float* __restrict a, const float* __restrict b, float* __restrict c, int rows, int columns, int leftColumnCount)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val;
			    if(j >= leftColumnCount)
				    val = b[(j-leftColumnCount) * rows + i];
			    else
				    val = a[j * rows + i];
			    c[j * rows + i] = val;
            }
        }
	}

	__global__ void EuclideanDistance(const float* __restrict a, const float* __restrict b, float* __restrict c, int count)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            c[index] = pow(a[index] - b[index], 2);
        }
	}

	/*__global__ void MultiEuclideanDistance(const float* __restrict a, const float* __restrict* b, float* __restrict c, int size, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val1 = a[i];
			    float val2 = b[j][i];
			    c[j * size + i] = pow(val1 - val2, 2);
            }
        }
	}*/

	__global__ void ManhattanDistance(const float* __restrict a, const float* __restrict b, float* __restrict c, int count)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            c[index] = abs(a[index] - b[index]);
        }
	}

	/*__global__ void MultiManhattanDistance(const float* __restrict a, const float* __restrict* b, float* __restrict c, int size, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val1 = a[i];
			    float val2 = b[j][i];
			    c[j * size + i] = abs(val1 - val2);
            }
        }
	}*/

	__global__ void CosineDistance(const float* __restrict a, const float* __restrict b, float* __restrict aa, float* __restrict ab, float* __restrict bb, int count)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
			float left = a[index];
			float right = b[index];
			atomicAdd(aa, left * left);
			atomicAdd(ab, left * right);
			atomicAdd(bb, right * right);
        }
	}

	__global__ void Abs(const float* __restrict a, float* __restrict b, int count)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index] = abs(a[index]);
        }
	}

	__global__ void Log(const float* __restrict a, float* __restrict b, int count)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index] = log(a[index]);
        }
	}

	__global__ void Normalise(float* __restrict a, int count, float min, float range)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            a[index] = (a[index] - min) / range;
        }
	}

	__global__ void SoftmaxVector(const float* __restrict a, float* __restrict b, int count, float max)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < count; index += blockDim.x * gridDim.x) {
            b[index] = exp(a[index] - max);
        }
	}

	__global__ void VectorAdd(float* __restrict a, int size, float scalar)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            a[index] += scalar;
        }
	}

	__global__ void VectorCopyRandom(const float* __restrict a, float* __restrict b, int* __restrict c, int size)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
			b[index] += a[c[index]];
        }
	}

	__global__ void CopyToMatrixRows(const float* __restrict* a, float* __restrict b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[i][j];
			    b[j * rows + i] = val;
            }
        }
	}

    __global__ void CopyToMatrixColumns(const float* __restrict* a, float* __restrict b, int rows, int columns)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < rows; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                float val = a[j][i];
                //printf("i:%i(%i) j:%i(%i)\n", i, rows, j, columns);
			    b[j * rows + i] = val;
            }
        }
	}

	__global__ void TensorAddPadding(
        int size, 
        const float* __restrict a, 
        float* __restrict b, 
        int rows, 
        int columns, 
        int depth, 
        int count, 
        int outputRows, 
        int outputColumns, 
        int padding
    ) {
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int i = index % outputRows;
            int index2 = index / outputRows;

            int j = index2 % outputColumns;
            int index3 = index2 / outputColumns;

            int k = index3 % depth;
            int z = index3 / depth;

            float val = 0;
            if(i >= padding && i < (outputRows - padding) && j >= padding && j < (outputColumns - padding)) {
                const float* inputPtr = a + (rows * columns * depth * z) + (rows * columns * k);
                int aIndex = (j-padding) * rows + (i-padding);
                val = inputPtr[aIndex];

                /*printf("index:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) ai:%i val:%f\n", index,
                    i, outputRows,
                    j, outputColumns, 
                    k, depth, 
                    z, count,
                    aIndex, val
                );*/
            }

            float* outputPtr = b + (outputRows * outputColumns * depth * z) + (outputRows * outputColumns * k);
            outputPtr[j * outputRows + i] = val;
        }
	}

	__global__ void TensorRemovePadding(
        int size, 
        const float* __restrict a, 
        float* __restrict b, 
        int rows, 
        int columns, 
        int depth, 
        int count, 
        int outputRows, 
        int outputColumns, 
        int padding
    ) {
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int i = index % rows;
            int index2 = index / rows;

            int j = index2 % columns;
            if(i >= padding && i < (rows-padding) && j >= padding && j < (columns-padding)) {
                int index3 = index2 / columns;

                int k = index3 % depth;
                int z = index3 / depth;

                const float* inputPtr = a + (rows * columns * depth * z) + (rows * columns * k);
                int aIndex = j * rows + i;
                float val = inputPtr[aIndex];

                float* outputPtr = b + (outputRows * outputColumns * depth * z) + (outputRows * outputColumns * k);
                int bIndex = (j-padding) * outputRows + (i-padding);
                outputPtr[bIndex] = val;

                /*printf("index:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) ai:%i bi:%i val:%f\n", index,
                    i, outputRows,
                    j, outputColumns, 
                    k, depth, 
                    z, count,
                    aIndex, bIndex,
                    val
                );*/
            }
        }
	}

    __global__ void TensorIm2Col(
        int size, 
        const float* __restrict a, 
        float* __restrict b, 
        const float* __restrict cx, 
        const float* __restrict cy, 
        int rows,
        int columns,
        int depth,
        int count,
        int outputRows,
        int outputColumns,
        int convolutionCount, 
        int filterWidth, 
        int filterHeight,
        int xStride,
		int yStride
    ) {
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int x = index % filterWidth;
            int index2 = index / filterWidth;

            int y = index2 % filterHeight;
            int index3 = index2 / filterHeight;

            int k = index3 % depth;
            int index4 = index3 / depth;

            int ci = index4 % convolutionCount;
            int i = index4 / convolutionCount;

            int offsetX = cx[ci];
            int offsetY = cy[ci];

            /*printf("index:%i, i:%i(%i), ci:%i(%i), k:%i(%i), x:%i(%i), y:%i(%i), cx:%i=%i, cy:%i=%i\n", index,
                i, count,
                ci, convolutionCount,
                k, depth,
                x, filterWidth,
                y, filterHeight,
                offsetX, (int)cx[ci],
                offsetY, (int)cy[ci]
            );*/

            int filterOffset = k * filterWidth * filterHeight;
            int filterIndex = filterOffset + (x * filterHeight + y);

            float* outputPtr = b + (outputRows * outputColumns * i);
            const float* inputPtr = a + (rows * columns * depth * i) + (rows * columns * k);
            outputPtr[filterIndex * outputRows + ci] = inputPtr[(offsetX + x) * rows + (offsetY + y)];
        }
    }

    __global__ void TensorReverseIm2Col(
        int size, 
        const float* __restrict a, 
        const float* __restrict filters, 
        float* __restrict b, 
        const float* __restrict cx, 
        const float* __restrict cy, 
        int rows, 
        int columns, 
        int depth, 
        int count,
        int convolutionCount,  
        int filterWidth, 
        int filterHeight, 
        int xStride,
		int yStride,
        int outputRows,
        int outputColumns,
        int outputDepth
    ) {
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int z = index % outputDepth;
            int index2 = index / outputDepth;

            int x = index2 % filterWidth;
            int index3 = index2 / filterWidth;

            int y = index3 % filterHeight;
            int index4 = index3 / filterHeight;

            int ci = index4 % convolutionCount;
            int index5 = index4 / convolutionCount;

            int k = index5 % depth;
            int i = index5 / depth;

            int offsetX = cx[ci];
            int offsetY = cy[ci];

            /*printf("index:%i di:%i(%i) ci:%i(%i) k:%i(%i) x:%i(%i) y:%i(%i) z:%i(%i) cx:%i cy:%i\n", index,
                i, count,
                ci, convolutionCount, 
                k, depth, 
                x, filterWidth, 
                y, filterHeight, 
                z, outputDepth, 
                offsetX, offsetY
            );*/

            const float* slice = a + (i * rows * columns * depth) + (k * rows * columns);
            const float* filter = filters + (k * outputDepth * filterWidth * filterHeight) + (z * filterWidth * filterHeight);
            float* output = b + (i * outputRows * outputColumns * outputDepth) + (z * outputRows * outputColumns);

            int errorX = offsetX / xStride;
            int errorY = offsetY / yStride;
            if(errorX < columns && errorY < rows) {
                float error = slice[errorX * rows + errorY];

                int filterIndex = (filterWidth-x-1) * filterHeight + (filterHeight-y-1);
                int outputIndex = (offsetX+x) * outputRows + (offsetY+y);
                float val = filter[filterIndex] * error;

                atomicAdd(output + outputIndex, val);
            }
        }
    }

	__global__ void SoftmaxDerivative(const float* __restrict a, float* __restrict b, int size)
	{
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < size; j += blockDim.y * gridDim.y) {
                int index = j * size + i;
			    if(i == j)
				    b[index] = a[i] * (1 - a[i]);
			    else
				    b[index] = -a[i] * a[j];
            }
        }
	}

	__global__ void RotateInPlace(float* __restrict a, int size, int blockCount, int blockSize)
	{
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int blockIndex = index / blockSize;
			int blockOffset = index % blockSize;
            int index1 = blockIndex * blockSize + blockSize - blockOffset - 1;
			int index2 = blockIndex * blockSize + blockOffset; 
			float temp = a[index1];
			a[index1] = a[index2];
			a[index2] = temp;
        }
	}

	__global__ void TensorMaxPool(
        int size, 
        const float* __restrict a, 
        float* __restrict b, 
        float* __restrict indexOffset,
        const float* __restrict cx, 
        const float* __restrict cy,
        int convolutionCount,
        int rows, 
        int columns, 
        int depth, 
        int count, 
        int outputRows, 
        int outputColumns, 
        int filterWidth, 
        int filterHeight, 
        int xStride,
		int yStride,
        int saveIndices
    ) {
		for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int ci = index % convolutionCount;
            int index2 = index / convolutionCount;

            int k = index2 % depth;
            int z = index2 / depth;

            int aX = cx[ci];
			int aY = cy[ci];
            int bX = aX / xStride;
            int bY = aY / yStride;

            /*printf("index:%i k:%i(%i) z:%i(%i) ax:%i ay:%i bx:%i by:%i\n", index,
                k, depth, 
                z, count,
                aX, aY,
                bX, bY
            );*/

            int targetOffset = (z * outputRows * outputColumns * depth) + (k * outputRows * outputColumns);
            const float* source = a + (z * rows * columns * depth) + (k * rows * columns);
            float* target = b + targetOffset;

            float maxVal = 0;
	        int bestOffset = -1;
	        int offset = 0;
	                
	        for (int x = 0; x < filterWidth; x++) {
		        for (int y = 0; y < filterHeight; y++) {
			        float val = source[(aX + x) * rows + (aY + y)];
                    bool isGreater = (bestOffset < 0 || val > maxVal);
			        if (isGreater) {
				        bestOffset = offset;
				        maxVal = val;
			        }
                    //printf("index:%i, x:%i, y:%i val:%f max:%f offset:%i is-greater:%i\n", index, x, y, val, maxVal, bestOffset, isGreater ? 1 : 0);
					++offset;
		        }
	        }

            //printf("\tindex:%i i:%i j:%i val:%f\n", index, i, j, maxVal);
            if(saveIndices) {
                float* indices = indexOffset + targetOffset;
                indices[bX * outputRows + bY] = bestOffset;
            }
            target[bX * outputRows + bY] = maxVal;
        }
	}

	__global__ void TensorReverseMaxPool(
        int size, 
        const float* __restrict a,
        const float* __restrict indices,
        float* __restrict b, 
        int rows,
        int columns,
        int depth,
        int count,
        int outputRows,
        int outputColumns,
        int filterWidth,
        int filterHeight,
        int xStride,
		int yStride
    ) {
        for (int index = blockDim.x * blockIdx.x + threadIdx.x; index < size; index += blockDim.x * gridDim.x) {
            int i = index % rows;
            int index2 = index / rows;

            int j = index2 % columns;
            int index3 = index2 / columns;

            int k = index3 % depth;
            int z = index3 / depth;

            int sourceOffset = (z * rows * columns * depth) + (k * rows * columns);
            const float* source = a + sourceOffset;
            const float* indexPtr = indices + sourceOffset;
            float* target = b + (z * outputRows * outputColumns * depth) + (k * outputRows * outputColumns);
            int sourceIndex = j * rows + i;
            float val = source[sourceIndex];
            int offset = indexPtr[sourceIndex];

            if(offset < 0)
                offset = 0;

            int targetX = j * xStride + (offset / filterHeight);
            int targetY = i * yStride + (offset % filterHeight);

            /*printf("index:%i s:%i i:%i(%i) j:%i(%i) k:%i(%i) z:%i(%i) val:%f offset:%i tx:%i ty:%i\n", 
                index, xStride, yStride
                i, outputRows,
                j, outputColumns, 
                k, depth, 
                z, count,
                val, offset,
                targetX, targetY
            );*/

            target[targetX * outputRows + targetY] = val;
        }
	}

    __global__ void CalculateDistances(
        const float** __restrict a,
        const float** __restrict b,
        float* __restrict c,
        int rows,
        int columns,
        int size,
        int distanceMetric
    ) {
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                for (int k = blockDim.z * blockIdx.z + threadIdx.z; k < rows; k += blockDim.z * gridDim.z) {
                    float aVal = a[j][i];
                    float bVal = b[k][i];
                    float output = 0;

                    if(distanceMetric == 0) { // euclidean
                        float diff = aVal - bVal;
                        output = diff * diff;
                    }else if(distanceMetric == 1) { // cosine
                        output = aVal * bVal;
                    }else if(distanceMetric == 2) { // manhattan
                        output = abs(aVal - bVal);
                    }
                    float* outputPtr = c + (j * rows + k);
                    atomicAdd(outputPtr, output);
                }
            }
        }
	}

	__global__ void MultiCosineDistance(
		const float** __restrict a, 
		const float** __restrict b, 
		float* __restrict aa, 
		float* __restrict ab, 
		float* __restrict bb, 
		int rows,
        int columns,
        int size
	) {
        for (int i = blockDim.x * blockIdx.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x) {
            for (int j = blockDim.y * blockIdx.y + threadIdx.y; j < columns; j += blockDim.y * gridDim.y) {
                for (int k = blockDim.z * blockIdx.z + threadIdx.z; k < rows; k += blockDim.z * gridDim.z) {
					float aVal = a[j][i];
					float bVal = b[k][i];
					int offset = j * rows + k;
					atomicAdd(aa + offset, aVal * aVal);
					atomicAdd(ab + offset, aVal * bVal);
					atomicAdd(bb + offset, bVal * bVal);
				}
            }
        }
	}
}